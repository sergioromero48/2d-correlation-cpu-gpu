#include <stdio.h>
#include <hip/hip_runtime.h>

#define INPUT_MATRIX_SIZE 256  // 256x256 input matrix
#define KERNEL_SIZE 8          // 8x8 kernel 
#define OUTPUT_MATRIX_SIZE (INPUT_MATRIX_SIZE - KERNEL_SIZE + 1) // output matrix

__constant__ int kernel[KERNEL_SIZE * KERNEL_SIZE]; // creates kernel

__global__ void cross_correlation_2d(float* input, float* output) {

    int x = blockIdx.x * blockDim.x + threadIdx.x; // Column
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Row

    if (x < OUTPUT_MATRIX_SIZE && y < OUTPUT_MATRIX_SIZE) {
        float sum = 0.0f;
        // Correlation Calculation
        for (int m = 0; m < KERNEL_SIZE; ++m) {
            for (int n = 0; n < KERNEL_SIZE; ++n) {
                int input_row = y + m;
                int input_col = x + n;
                float input_val = input[input_row * INPUT_MATRIX_SIZE + input_col]; 
                float kernel_val = kernel[m * KERNEL_SIZE + n];
                sum += input_val * kernel_val;
            }
        }
        // output matrix
        output[y * OUTPUT_MATRIX_SIZE + x] = sum;
    }
}

int main() {
    // Host matrices
    float h_input[INPUT_MATRIX_SIZE][INPUT_MATRIX_SIZE], h_kernel[KERNEL_SIZE][KERNEL_SIZE], h_output[OUTPUT_MATRIX_SIZE][OUTPUT_MATRIX_SIZE];
    float *d_input, *d_output;

    // Initialize matrices with random sample values
    for (int i = 0; i < INPUT_MATRIX_SIZE; i++) {
        for (int j = 0; j < INPUT_MATRIX_SIZE; j++) {
            h_input[i][j] = (2.0f * rand()/RAND_MAX - 1.0f);  // Sample input
        }
    }

    for (int i = 0; i < KERNEL_SIZE; i++){
        for (int j = 0; j < KERNEL_SIZE; j++){
            h_kernel[i][j] = (2.0f * rand()/RAND_MAX - 1.0f);  // Sample kernel
        }
    }
    

    // Allocate device memory
    hipMalloc(&d_input, INPUT_MATRIX_SIZE * INPUT_MATRIX_SIZE * sizeof(float));
    hipMalloc(&d_output, OUTPUT_MATRIX_SIZE * OUTPUT_MATRIX_SIZE * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, h_input, INPUT_MATRIX_SIZE * INPUT_MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    
    // Copy Const kernel to device
    hipMemcpyToSymbol(HIP_SYMBOL(kernel), h_kernel, KERNEL_SIZE * KERNEL_SIZE*sizeof(float));

    // Create Events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start Measuring
    hipEventRecord(start);

    // Launch kernel
    dim3 threadsPerBlock(16, 16);
    dim3 Blocks((OUTPUT_MATRIX_SIZE + 15) / 16, (OUTPUT_MATRIX_SIZE + 15) / 16);
    cross_correlation_2d<<<Blocks, threadsPerBlock>>>(d_input, d_output);

    // Stop Measuring 
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate Elapsed Time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Output Time Duration
    printf("TIME ELAPSED: %f MILLISECONDS\n", milliseconds);

    // Copy result back to host
    hipMemcpy(h_output, d_output, INPUT_MATRIX_SIZE * INPUT_MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_input);
    hipFree(kernel);
    hipFree(d_output);

    printf("COMPLETED SUCCESSFULLY!\n");

    return 0;
}
